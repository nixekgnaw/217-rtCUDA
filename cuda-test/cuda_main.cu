#include "hip/hip_runtime.h"
// smallptCUDA by Sam Lapere, 2015
// based on smallpt, a path tracer by Kevin Beason, 2008  

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include ""
#include "cutil_math.h" // from http://www.icmc.usp.br/~castelo/CUDA/common/inc/cutil_math.h

#define M_PI 3.14159265359f  // pi
#define width 512  // screenwidth
#define height 384 // screenheight
#define samps 200 // samples 

// __device__ : executed on the device (GPU) and callable only from the device


struct Ray {
    float3 o,d; // ���ߵ���ʼ�ͷ��� ray origin & direction 
    __device__ Ray(float3 o_, float3 d_) : o(o_), d(d_) {}
};

enum Refl_t 
{
    DIFF, 
    SPEC, 
    REFR };  // material types, used in radiance(), ��ǰֻ�������� ����֮��Ҫ��Ҫ�� only DIFF used here

struct Sphere {

    float rad;            // radius ��뾶
    float3 p, e, c; // ��Բ�� e? ��ɫposition, emission, colour 
    Refl_t refl;          // ���� reflection type (e.g. diffuse)
    //__device__ Sphere(float rad_, float3 p_, float3 e_, float3 c_, Refl_t refl_) : rad(rad_), p(p_), e(e_), c(c_), refl(refl_) {} 
    __device__ float intersect(const Ray& r) const 
    { // returns distance, 0 if nohit
        float3 op = p - r.o;    //��һԪ���η��� ������ѧ Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0
        float t, eps = 0.0001f;  //����try 1e-4 epsilon required to prevent floating point precision artefacts
        float b = dot(op,r.d);    // b ��������
        float det = b * b - dot(op, op) + rad * rad;  // ������ѧdiscriminant quadratic equation
        if (det < 0) 
            return 0;       // ������ѧ
        else 
            det = sqrtf(det);    // sqrtf�� sqrt����н����ж�������
        return (t = b - det) > eps ? t : ((t = b + det) > eps ? t : 0); // ȡclosest one
    }
};

// SCENE
// 9���� 9 spheres forming a Cornell box
// �����Ż�,����˵����==�ó����ڴ���Ⱦ��small enough to be in constant GPU memory
// { float radius, { float3 position }, { float3 emission }, { float3 colour }, refl_type }
__constant__ Sphere spheres[] = {
 { 1e5f, { 1e5f + 1.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.25f, 0.25f }, DIFF }, //Left 
 { 1e5f, { -1e5f + 99.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .25f, .25f, .75f }, DIFF }, //Rght 
 { 1e5f, { 50.0f, 40.8f, 1e5f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Back 
 { 1e5f, { 50.0f, 40.8f, -1e5f + 600.0f }, { 0.0f, 0.0f, 0.0f }, { 1.00f, 1.00f, 1.00f }, DIFF }, //Frnt 
 { 1e5f, { 50.0f, 1e5f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Botm 
 { 1e5f, { 50.0f, -1e5f + 81.6f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Top 
 { 16.5f, { 27.0f, 16.5f, 47.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, DIFF }, // small sphere 1
 { 16.5f, { 73.0f, 16.5f, 78.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, DIFF }, // small sphere 2
 { 600.0f, { 50.0f, 681.6f - .77f, 81.6f }, { 2.0f, 1.8f, 1.6f }, { 0.0f, 0.0f, 0.0f }, DIFF }  // Light
};

//__device__ inline float clamp(float x) { return x < 0 ? 0 : x > 1 ? 1 : x; }
//__device__ inline int toInt(float x) { return int(pow(clamp(x), 1 / 2.2) * 255 + .5); }
__device__ inline bool intersect_scene(const Ray& r, float& t, int& id)
{
    float n = sizeof(spheres) / sizeof(Sphere), d, inf = t = 1e20;  // t is distance to closest intersection, initialise t to a huge number outside scene
    for (int i = int(n); i--;)  // test all scene objects for intersection
        if ((d = spheres[i].intersect(r)) && d < t) 
        {  // if newly computed intersection distance d is smaller than current closest intersection distance
            t = d;  // keep track of distance along ray to closest intersection point 
            id = i; // and closest intersected object
        }
    return t < inf; // returns true if an intersection with the scene occurred, false when no hit
}

// !!����ֵֹ����������==
//random number generator from https://github.com/gz/rust-raytracer
__device__ static float getrandom(unsigned int* seed0, unsigned int* seed1) {
    *seed0 = 36969 * ((*seed0) & 65535) + ((*seed0) >> 16);  // hash the seeds using bitwise AND and bitshifts
    *seed1 = 18000 * ((*seed1) & 65535) + ((*seed1) >> 16);

    unsigned int ires = ((*seed0) << 16) + (*seed1);

    // Convert to float
    union {
        float f;
        unsigned int ui;
    } res;

    res.ui = (ires & 0x007fffff) | 0x40000000;  // bitwise AND, bitwise OR

    return (res.f - 2.f) / 2.f;
}

// radiance function, the meat of path tracing 
// solves the rendering equation: 
// outgoing radiance (at a point) = emitted radiance + reflected radiance
// reflected radiance is sum (integral) of incoming radiance from all directions in hemisphere above point, 
// multiplied by reflectance function of material (BRDF) and cosine incident angle 
__device__ float3 radiance(Ray& r, unsigned int* s1, unsigned int* s2) { // returns ray color

    float3 accucolor = make_float3(0.0f, 0.0f, 0.0f); // accumulates ray colour with each iteration through bounce loop
    float3 mask = make_float3(1.0f, 1.0f, 1.0f);

    // ray bounce loop (no Russian Roulette used) 
    for (int bounces = 0; bounces < 4; bounces++)
    {  // ��������ѭ�����ǵݹ�,����û�ö���˹���̴��ġ�����������Եݹ��Ч��(replaces recursion in CPU code)
        float t;           // distance to intersection
        int id = 0;        // id of intersected object
        if (!intersect_scene(r, t, id))
            return make_float3(0.0f, 0.0f, 0.0f); // û�򵽷��غ�ɫif miss, return black
        const Sphere& obj = spheres[id];  // the hit object
        float3 x = r.o + r.d * t;          // ���� hitpoint 
        float3 n = normalize(x - obj.p);    // ���� normal
        float3 nl = dot(n, r.d) < 0 ? n : n * -1; // ������Զ�����䷴����

        //����˹���̶ģ��Ĵ���
        //float p = f.x > f.y && f.x > f.z ? f.x : f.y > f.z ? f.y : f.z; // max refl
        //if (++depth > 5)
        //    if (hiprand_uniform(rand_state) < p)
        //        f = f * (1 / p);
        //    else
        //        return obj.e; //R.R.

        // add emission of current sphere to accumulated colour
        // (first term in rendering equation sum) 
        accucolor += mask * obj.e;
        if (obj.refl == DIFF)
        {
      
            float r1 = 2 * M_PI * getrandom(s1, s2); // ȡһ�������
            float r2 = getrandom(s1, s2);  // ȡ�ڶ��������
            float r2s = sqrtf(r2);
            //���ּ���
            float3 w = nl;
            float3 u = normalize(cross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
            float3 v = cross(w, u);
            float3 d = normalize(u * cos(r1) * r2s + v * sin(r1) * r2s + w * sqrtf(1 - r2));

            // new ray origin is intersection point of previous ray with scene
            r.o = x + nl * 0.05f; // offset ray origin slightly to prevent self intersection
            r.d = d;
            mask *= obj.c;    // multiply with colour of object       
            mask *= dot(d, nl);  // weigh light contribution using cosine of angle between incident light and normal
            mask *= 2;          // fudge factor
        }
    }

    return accucolor;
}


// __global__ : executed on the device (GPU) and callable only from host (CPU) 
// this kernel runs in parallel on all the CUDA threads

__global__ void render_kernel(float3* output) {

    // assign a CUDA thread to every pixel (x,y) 
    // blockIdx, blockDim and threadIdx are CUDA specific keywords
    // replaces nested outer loops in CPU code looping over image rows and image columns 
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int i = (height - y - 1) * width + x; // index of current pixel (calculated using thread index) 

    unsigned int s1 = x;  // seeds for random number generator
    unsigned int s2 = y;

    // generate ray directed at lower left corner of the screen
    // compute directions for all other rays by adding cx and cy increments in x and y direction
    Ray cam(make_float3(50, 52, 295.6), normalize(make_float3(0, -0.042612, -1))); // first hardcoded camera ray(origin, direction) 
    float3 cx = make_float3(width * .5135 / height, 0.0f, 0.0f); // ray direction offset in x direction
    float3 cy = normalize(cross(cx, cam.d)) * .5135; // ray direction offset in y direction (.5135 is field of view angle)
    float3 r; // r is final pixel color       

    r = make_float3(0.0f); // reset r to zero for every pixel 

    for (int s = 0; s < samps; s++) {  // samples per pixel

     // compute primary ray direction
        float3 d = cam.d + cx * ((.25 + x) / width - .5) + cy * ((.25 + y) / height - .5);

        // create primary ray, add incoming radiance to pixelcolor
        r = r + radiance(Ray(cam.o + d * 40, normalize(d)), &s1, &s2) * (1. / samps);
    }       // Camera rays are pushed ^^^^^ forward to start in interior 

    // write rgb value of pixel to image buffer on the GPU, clamp value to [0.0f, 1.0f] range
    output[i] = make_float3(clamp(r.x, 0.0f, 1.0f), clamp(r.y, 0.0f, 1.0f), clamp(r.z, 0.0f, 1.0f));
}

inline float clamp(float x) { return x < 0.0f ? 0.0f : x > 1.0f ? 1.0f : x; }

inline int toInt(float x) { return int(pow(clamp(x), 1 / 2.2) * 255 + .5); }  // convert RGB float in range [0,1] to int in range [0, 255] and perform gamma correction

int main() {

    float3* output_h = new float3[width * height]; // pointer to memory for image on the host (system RAM)
    float3* output_d;    // pointer to memory for image on the device (GPU VRAM)

    // allocate memory on the CUDA device (GPU VRAM)
    hipMalloc(&output_d, width * height * sizeof(float3));

    // dim3 is CUDA specific type, block and grid are required to schedule CUDA threads over streaming multiprocessors
    dim3 block(8, 8, 1);
    dim3 grid(width / block.x, height / block.y, 1);

    printf("CUDA initialised.\nStart rendering...\n");

    // schedule threads on device and launch CUDA kernel from host
    render_kernel << < grid, block >> > (output_d);

    // copy results of computation from device back to host
    hipMemcpy(output_h, output_d, width * height * sizeof(float3), hipMemcpyDeviceToHost);

    // free CUDA memory
    hipFree(output_d);

    printf("Done!\n");

    // Write image to PPM file, a very simple image file format
    FILE* f = fopen("smallptcuda.ppm", "w");
    fprintf(f, "P3\n%d %d\n%d\n", width, height, 255);
    for (int i = 0; i < width * height; i++)  // loop over pixels, write RGB values
        fprintf(f, "%d %d %d ", toInt(output_h[i].x),
            toInt(output_h[i].y),
            toInt(output_h[i].z));

    printf("Saved image to 'smallptcuda.ppm'\n");

    delete[] output_h;
    system("PAUSE");
}